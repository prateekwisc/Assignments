#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<sys/time.h>

__global__
void Matadd(char* A,int N)
 {
      
 }

int main()
{     

for(int j=0;j<=30;j++)
{     
      
      hipEvent_t start1,stop1; 
      float time1;
      int i;
      int N = pow(2,j);
      size_t size = N;
printf ("\n The value of N is %d",N);

hipEventCreate(&start1);
hipEventCreate(&stop1);     



//pinned memory allocate in host memory
char* hA; hipHostMalloc(&hA, size, hipHostMallocDefault);


for(i=0;i<N;i++)
{
hA[i] = rand()%20-10;

 }
//allocate memory on the device (GPU)
char* dA;
hipMalloc((void**) &dA,size);

//timing start for inclusive timing
hipEventRecord(start1, 0);
     
//copy vectors from host memory to devie memory
      hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);

hipEventRecord(stop1, 0);

hipEventSynchronize(stop1);


//invoke GPU kernel, with two blocks each having eight threads

      
int threadsperblock = 16;
int blockspergrid = (N + threadsperblock - 1)/ threadsperblock;

//timing start for exclusive timing
//cudaEventRecord(start2, 0);
Matadd<<<blockspergrid,threadsperblock>>>(dA,N);

hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost);



//cudaEventRecord(stop1, 0);
     
//cudaEventSynchronize(stop1);
hipEventElapsedTime(&time1,start1,stop1);
      

printf("\n The data transfer time(in microsecs) for 2 to power %d is %f respectively \n",j,time1);



hipFree(hA);

hipFree(dA);

}
return 0;
}
