#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<sys/time.h>
#include<stdlib.h>
__global__
void Matadd(char* A,int N)
 {
      
 }

int main()
{     

  char newline = '\n';
  
FILE *fp;
fp = fopen("HW7_1-1.txt","w");

for(int j=0;j<=25;j++)
{     
      
      hipEvent_t start1,stop1; 
      float time1;
      int i;
      int N = pow(2,j);
      size_t size = N;
printf ("\n The value of N is %d",N);

hipEventCreate(&start1);
hipEventCreate(&stop1);     



//allocate input matrices hA, hB, hC,refC in host memory
char* hA = (char*)malloc(size);


for(i=0;i<N;i++)
{
hA[i] = rand()%20-10;

 }
//allocate memory on the device (GPU)
char* dA;
hipMalloc((void**) &dA,size);

//timing start for inclusive timing
hipEventRecord(start1, 0);
     
//copy vectors from host memory to devie memory
      hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);

        
hipEventRecord(stop1, 0);

hipEventSynchronize(stop1);


//invoke GPU kernel, with two blocks each having eight threads

      
int threadsperblock = 16;
int blockspergrid = (N + threadsperblock - 1)/ threadsperblock;

//timing start for exclusive timing
//cudaEventRecord(start2, 0);
Matadd<<<blockspergrid,threadsperblock>>>(dA,N);

hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost);



//cudaEventRecord(stop1, 0);
     
//cudaEventSynchronize(stop1);
hipEventElapsedTime(&time1,start1,stop1);
      

printf("\n The transfer time in microseconds for 2 to power %d is %f respectively \n",j,time1 );

fwrite(&j,sizeof(j),1,fp);
fwrite(&time1,sizeof(time1),1,fp);
fwrite(&newline,sizeof(newline),1,fp);


hipFree(hA);

hipFree(dA);

}
fclose(fp);

return 0;
}
