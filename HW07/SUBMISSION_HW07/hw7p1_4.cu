#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<sys/time.h>

__global__
void Matadd(char* A, char*B, int N)
 {
      
 }

int main()
{     

for(int j=0;j<=25;j++)
{     
      
      hipEvent_t start1,stop1,start2,stop2; 
      float time1,time2, time3;
      int i;
      int N = pow(2,j);
      size_t size = N;
printf ("\n The value of N is %d",N);

hipEventCreate(&start1);
hipEventCreate(&stop1);     

hipEventCreate(&start2);
hipEventCreate(&stop2);     


//allocate input matrices hA, hB, hC,refC in host memory
char* hA; hipHostMalloc(&hA, size, hipHostMallocDefault);
char* hB; hipHostMalloc(&hB, size, hipHostMallocDefault);

for(i=0;i<N;i++)
{
hA[i] = rand()%20-10;

 }
//allocate memory on the device at location A (GPU)
char* dA;
hipMalloc((void**) &dA,size);


//allocate memory on the device at location B (GPU)
char* dB;
hipMalloc((void**) &dB,size);

//timing start for inclusive timing
hipEventRecord(start1, 0);
     
//copy vectors from host memory to devie memory
      hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);


hipEventRecord(stop1, 0);

hipEventSynchronize(stop1);

//invoke GPU kernel, with two blocks each having eight threads

      
int threadsperblock = 16;
int blockspergrid = (N + threadsperblock - 1)/ threadsperblock;


hipEventRecord(start2, 0);

//timing start for exclusive timing
//cudaEventRecord(start2, 0);
Matadd<<<blockspergrid,threadsperblock>>>(dA,dB,N);


hipMemcpy(hB, dB, size, hipMemcpyDeviceToHost);


hipEventRecord(stop2, 0);
     
hipEventSynchronize(stop2);

hipEventElapsedTime(&time1,start1,stop1);
      
hipEventElapsedTime(&time2,start2,stop2);

printf("\n The Host to Device time  for location A in microseconds for 2 to power %d is %f respectively \n",j,time1);

printf("\n The Device to Host time  for location B in microseconds for 2 to power %d is %f respectively \n",j,time2);

time3 = time1 + time2;

printf("\n The total data transfer time  in microseconds for 2 to power %d is %f respectively \n",j,time3);


hipFree(hA);
hipFree(hB);

hipFree(dA);
hipFree(dB);
}
return 0;
}
