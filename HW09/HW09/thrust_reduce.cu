#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/reduce.h>
#include<thrust/scan.h>
#include<>
#include<stdio.h>
#include<stdlib.h>
#define DATA_SIZE 66292994
int main(void)
{
 
  float t_reduce;
  hipEvent_t start1,stop1, start2, stop2;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  //initialze random values on host
  thrust::host_vector<int> data(DATA_SIZE);
  
  thrust::generate(data.begin(), data.end(), rand);



//compute sum on host(CPU)
int h_sreduce = thrust::reduce(data.begin(), data.end());


//for inclusive time
hipEventRecord(start1, NULL);
//copy values on device
  thrust::device_vector<int> gpudata = data;




hipEventRecord(start2, NULL);
//compute sum on device(GPU)  
  int d_sreduce = thrust::reduce(gpudata.begin(), gpudata.end());
//copy back to host
  thrust::copy(gpudata.begin(), gpudata.end(), data.begin());

hipEventRecord(stop1, NULL);
hipEventSynchronize(stop1);
hipEventElapsedTime(&t_reduce, start1, stop1);
printf("\n Reduce time is %f ms", t_reduce);
// thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

printf("\n host sum = %d, gpu sum = %d",h_sreduce,d_sreduce);

}
