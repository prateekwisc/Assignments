#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<sys/time.h>


__global__
void Matmultkernel(int* A , int* b, int* C)
{
    
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int P = 0;

for(int k= 0; k<32; k++)
 {
  int MA = A[k + col*32];
  int Mb = b[k];
  P += MA * Mb;
 }
  C[col] = P;
  
}

int main()
{
int i,j;
    

int d_A[16][32];
int d_b[32][1];
int d_C[16][1]; 

for(i = 0;i<16;i++)
 {
   for (j=0;j<32;j++)
    {
     d_A[i][j] = i + j;
    printf(" %d ",d_A[i][j]);
    }
    printf("\n"); 
 }
printf("\n");

for(i=0;i<32;i++)
 {
   d_b[i][0] = i;
   printf(" %d \t", d_b[i][0]);
 }
 
size_t sizeA = 16 * 32 * sizeof(int);

size_t sizeb = 32 * sizeof(int);

size_t sizeC = 16 * sizeof(int);

int* A;
hipMalloc(&A,sizeA);

int* b;
hipMalloc(&b,sizeb);

int* C;
hipMalloc(&C,sizeC);

//Allocate and Load A and B into device memory


hipMemcpy(A, d_A, sizeA, hipMemcpyHostToDevice);

hipMemcpy(b, d_b, sizeb, hipMemcpyHostToDevice);


// Invoke kernel

Matmultkernel<<<1,16>>>(A, b, C);

//bring the result back from the device memory into the host 
hipMemcpy(d_C, C, sizeC, hipMemcpyDeviceToHost);


for(i = 0; i<16;i++)
{ 
  printf("\n %d", d_C[i][0]);
}



hipFree(A);
hipFree(b);
hipFree(C);

return 0;
}
