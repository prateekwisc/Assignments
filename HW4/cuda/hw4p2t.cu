#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>



__global__
void Matmultkernel(int* A , int* b, int* C)
{
    
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int P = 0;

for(int k= 0; k<32; k++)
 { 
  int MA = A[col * 32 + k];
  int Mb = b[k];
  P = P + MA * Mb;
 }
    
  C[col] = P;   
}

int main()
{
  FILE * pFile;
int i,j;

pFile = fopen("problem1.out","w");    

int d_A[16][32];
int d_b[32][1];
int d_C[16][1]; 
float time;

hipEvent_t start1, stop1;

hipEventCreate(&start1);

hipEventCreate(&stop1);

for(i = 0;i<16;i++)
 {
   for (j=0;j<32;j++)
    {
     d_A[i][j] = i + j;
    printf(" %d ",d_A[i][j]);
    }
 printf("\n");
 }
printf("\n");

 for(i=0; i<32; i++)
 {
   d_b[i][0] = i;
   printf(" %d \t", d_b[i][0]);
 }


size_t sizeA = 16 * 32 * sizeof(int);

size_t sizeb = 32 * sizeof(int);

size_t sizeC = 16 * sizeof(int);

int* A;
hipMalloc(&A,sizeA);

int* b;
hipMalloc(&b,sizeb);

int* C;
hipMalloc(&C,sizeC);

//Allocate and Load A and B into device memory


hipDeviceProp_t deviceProp;
const int currentDevice = 0;
if(hipGetDeviceProperties(&deviceProp, currentDevice) == hipSuccess)
  printf("Device %d: %s \n", currentDevice, deviceProp.name);


hipEventRecord(start1, 0);

hipMemcpy(A, d_A, sizeA, hipMemcpyHostToDevice);

hipMemcpy(b, d_b, sizeb, hipMemcpyHostToDevice);


// Invoke kernel

Matmultkernel<<<1,16>>>(A, b, C);

//bring the result back from the device memory into the host 
hipMemcpy(d_C, C, sizeC, hipMemcpyDeviceToHost);

hipEventRecord(stop1, 0);
hipEventSynchronize(stop1);

for(i = 0; i<16;i++)
{ 
  printf("\n %d", d_C[i][0]);
  fprintf(pFile, "%d \n",d_C[i][0]);
}

fclose (pFile);
hipFree(A);
hipFree(b);
hipFree(C);

hipEventElapsedTime(&time, start1, stop1);
printf("\n Inclusive time is %f", time);
hipEventDestroy(start1);
hipEventDestroy(stop1);

return 0;
}
