#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<sys/time.h>

__global__
void Matadd(double* A, double* B, double* C, int N)
 {
      
    int i = blockIdx.x* blockDim.x +threadIdx.x;
    if(i<N)
      C[i] = A[i] + B[i];
	__syncthreads();
 }

int main()
{     

for(int j=10;j<=20;j++)
{     
      
      hipEvent_t start1,start2,start3,stop1,stop2,stop3,start4,stop4; 
      float time1,time2,time3, time4;
      int i;
      int N = pow(2,j);
      size_t size = N * sizeof(double);
printf ("\n The value of N is %d",N);

hipEventCreate(&start1);
hipEventCreate(&stop1);     

hipEventCreate(&start2);
hipEventCreate(&stop2);     

hipEventCreate(&start3);
hipEventCreate(&stop3);     

hipEventCreate(&start4);
hipEventCreate(&stop4);     


//allocate input matrices hA, hB, hC,refC in host memory
double* hA = (double*)malloc(size);
double* hB = (double*)malloc(size);
double* hC = (double*)malloc(size);
double* refC = (double*)malloc(size);


for(i=0;i<N;i++)
{
hA[i] = rand()%20-10;
hB[i] = rand()%20-10;

refC[i] = hA[i] + hB[i];
 }
//allocate memory on the device (GPU)
double* dA;
hipMalloc(&dA,size);
double* dB;
hipMalloc(&dB,size);
double* dC;
hipMalloc(&dC,size);

//timing start for inclusive timing
hipEventRecord(start1, 0);
     
//copy vectors from host memory to devie memory
      hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);

      hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
        
//invoke GPU kernel, with two blocks each having eight threads

      
int threadsperblock = 32;
int blockspergrid = (N + threadsperblock - 1)/ threadsperblock;

//timing start for exclusive timing
hipEventRecord(start2, 0);
Matadd<<<blockspergrid,threadsperblock>>>(dA,dB,dC,N);
//timing stop for exclusive timing
hipEventRecord(stop2, 0);
hipEventSynchronize(stop2);
      
hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

//timing stop for inclusive timing
hipEventRecord(stop1, 0);
hipEventSynchronize(stop1);

     
//timing start for inclusive timing

hipEventRecord(start3, 0);


//copy vectors from host memory to devie memory
      hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);

      hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);


      
//invoke GPU kernel, with two blocks each having eight threads
threadsperblock = 1024;
blockspergrid = (N + threadsperblock - 1)/ threadsperblock;

//timing start for exclusive timing
hipEventRecord(start4, 0);
Matadd<<<blockspergrid,threadsperblock>>>(dA,dB,dC,N);
//timing stop for exclusive timing
hipEventRecord(stop4, 0);
hipEventSynchronize(stop4);

//bring the result back from the device memory into the host array
      hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

hipEventRecord(stop3, 0);
hipEventSynchronize(stop3);

for (i=0;i<N;i++)
{
if(fabs(refC[i] - hC[i]) > 1e-12)
{
printf("Erratic Value \n");
exit(1);
}
}

     
hipEventElapsedTime(&time1,start1,stop1);
      
hipEventElapsedTime(&time2,start2,stop2);

printf("\n The inclusive time and exclusive time for 32 threads in microseconds for 2 to power %d is %f and %f respectively \n",j,time1,time2);


hipEventElapsedTime(&time3,start3,stop3);

hipEventElapsedTime(&time4,start4,stop4);

printf("\n The inclusive time and exclusive time for 1024 threads in microseconds for 2 to power %d is %f and %f respectively \n",j,time3,time4);

free(hA);
free(hB);
free(hC);
free(refC);

hipFree(dA);
hipFree(dB);
hipFree(dC);

}
return 0;
}
