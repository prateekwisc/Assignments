#include<iostream>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 1000000
#define RADIUS 3

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.000001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeWeights(float* weights, int rad)
{
    // for now hardcoded for RADIUS=3
    weights[0] = 0.50f;
    weights[1] = 0.75f;
    weights[2] = 1.25f;
    weights[3] = 2.00f;
    weights[4] = 1.25f;
    weights[5] = 0.75f;
    weights[6] = 0.50f;
}

void initializeArray(float* arr, int nElements)
{
    const int myMinNumber = -5;
    const int myMaxNumber = 5;
    srand(time(NULL));
    for( int i=0; i<nElements; i++)
        arr[i] = (float)(rand() % (myMaxNumber - myMinNumber + 1) + myMinNumber);
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in, float *out) {
  
  for (int i = sIdx; i < eIdx; i++) {   
    out[i] = 0;
    //loop over all elements in the stencil
    for (int j = -RADIUS; j <= RADIUS; j++) {
      out[i] += weights[j + RADIUS] * in[i + j]; 
    }
    out[i] = out[i] / (2 * RADIUS + 1);
  }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float *weights, float *in, float *out) {
    int i = sIdx + blockIdx.x*blockDim.x + threadIdx.x;
    if( i < eIdx ) {
        float result = 0.f;
        result += weights[0]*in[i-3];
        result += weights[1]*in[i-2];
        result += weights[2]*in[i-1];
        result += weights[3]*in[i];
        result += weights[4]*in[i+1];
        result += weights[5]*in[i+2];
        result += weights[6]*in[i+3];
        result /=7.f;
        out[i] = result;
    }
}

int main() {
  int size = N * sizeof(float); 
  int wsize = (2 * RADIUS + 1) * sizeof(float); 
  //allocate resources
  float *weights = (float *)malloc(wsize);
  float *in      = (float *)malloc(size);
  float *out     = (float *)malloc(size); 
  float *cuda_out= (float *)malloc(size); 
  initializeWeights(weights, RADIUS);
  initializeArray(in, N);
  float *d_weights;  hipMalloc(&d_weights, wsize);
  float *d_in;       hipMalloc(&d_in, size);
  float *d_out;      hipMalloc(&d_out, size);
  
  hipMemcpy(d_weights,weights,wsize,hipMemcpyHostToDevice);
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  applyStencil1D<<<(N+511)/512, 512>>>(RADIUS, N-RADIUS, d_weights, d_in, d_out);
  applyStencil1D_SEQ(RADIUS, N-RADIUS, weights, in, out);
  hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);

  int nDiffs = checkResults(RADIUS, N-RADIUS, cuda_out, out);
  nDiffs==0? std::cout<<"Looks good.\n": std::cout<<"Doesn't look good: " << nDiffs << "differences\n";

  //free resources
  free(weights); free(in); free(out); free(cuda_out);
  hipFree(d_weights);  hipFree(d_in);  hipFree(d_out);
  return 0;
}
